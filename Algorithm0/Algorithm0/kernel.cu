#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//Todo: make this a template function
__device__
float* index2D(float* arr, int num_rows, int num_cols, int row_indx, int col_indx) {
    int index = (num_cols * row_indx) + col_indx;
    return &arr[index];
}

__device__
float max(float a, float b) {
    if (a < b) {
        return b;
    }
    else return a;
}

// prob_matrix is output
// haps is every haplotype, where each row is a haplotype
__global__ void lsf_kernel(float *prob_matrix, float *haps, float *gmap,
    const int target, const int num_snp, const int num_hap, const float read_error)
{
    __shared__ float buff0[num_hap];
    __shared__ float buff1[num_hap];

    float* currColumn = buff0;
    float* prevColumn = buff1;

    float probRead = 0;
    float probTrans = 0;
    float outLogLike = 0;
    int thread = threadIdx.x;

    // load first buffer
    prevColumn[thread] = 1 / num_hap;
    __syncthreads();

    // Do each column
    for (int snp_num = 1; snp_num < num_snp; snp_num++) {
        // calculate log likelihoods
        for (int hap_num = 0; hap_num < num_hap; hap_num++) {
            // don't compare target to itself, or it will always match itself
            if (hap_num != target) {
                // get emission probability
                if (*index2D(haps, num_hap, num_snp, target, snp_num)
                    == *index2D(haps, num_hap, num_snp, hap_num, snp_num))
                    probRead = 1 - read_error;
                else
                    probRead = read_error;

                // get transition probability
                if (target == hap_num)
                    probTrans = gmap[snp_num] - gmap[snp_num - 1];
                else
                    probTrans = 1 - (gmap[snp_num] - gmap[snp_num - 1]);

                // update current max probability explanation for observation
                outLogLike =
                    max(outLogLike, prevColumn[hap_num] + probTrans + probRead);
            }

            currColumn[thread] = outLogLike;
            __syncthreads();

            // copy curr column from shared memory out to global probability matrix
            *index2D(prob_matrix, num_hap, num_snp, snp_num, thread) = currColumn[thread];
            // swap prevColumn with currColumn in our circular buffer
            float* temp = prevColumn;
            prevColumn = currColumn;
            currColumn = temp;

            __syncthreads();
        }

        // sync threads
        __syncthreads();
    }
}

int main()
{
    // define data

    // allocate memory

    // copy memory

    // call kernel

    // copy memory back

    // print answer and matrix

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
