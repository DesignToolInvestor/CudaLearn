#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//Todo: make this a template function
__device__
float* index2D(float* arr, int num_rows, int num_cols, int row_indx, int col_indx) {
    int index = (num_cols * row_indx) + col_indx;
    return &arr[index];
}

__device__
float max(float a, float b) {
    if (a < b) {
        return b;
    }
    else return a;
}

// prob_matrix is output
// haps is every haplotype, where each row is a haplotype
__global__ void lsf_kernel(float *prob_matrix, float *haps, float *gmap,
    const int target, const int num_snp, const int num_hap, const float read_error)
{
    __shared__ float buff0[num_hap];
    __shared__ float buff1[num_hap];

    float* currColumn = buff0;
    float* prevColumn = buff1;

    float probRead = 0;
    float probTrans = 0;
    float outLogLike = 0;
    int thread = threadIdx.x;

    // load first buffer
    prevColumn[thread] = 1 / num_hap;
    __syncthreads();

    // Do each column
    for (int snp_num = 1; snp_num < num_snp; snp_num++) {
        // calculate log likelihoods
        for (int hap_num = 0; hap_num < num_hap; hap_num++) {
            // don't compare target to itself, or it will always match itself
            if (hap_num != target) {
                // get emission probability
                if (*index2D(haps, num_hap, num_snp, target, snp_num)
                    == *index2D(haps, num_hap, num_snp, hap_num, snp_num))
                    probRead = 1 - read_error;
                else
                    probRead = read_error;

                // get transition probability
                if (target == hap_num)
                    probTrans = gmap[snp_num] - gmap[snp_num - 1];
                else
                    probTrans = 1 - (gmap[snp_num] - gmap[snp_num - 1]);

                // update current max probability explanation for observation
                outLogLike =
                    max(outLogLike, prevColumn[hap_num] + probTrans + probRead);
            }

            currColumn[thread] = outLogLike;
            __syncthreads();

            // copy curr column from shared memory out to global probability matrix
            *index2D(prob_matrix, num_hap, num_snp, snp_num, thread) = currColumn[thread];
            // swap prevColumn with currColumn in our circular buffer
            float* temp = prevColumn;
            prevColumn = currColumn;
            currColumn = temp;

            __syncthreads();
        }

        // sync threads
        __syncthreads();
    }
}

int main()
{
    // define data


    // allocate memory

    // copy memory

    // call kernel

    // copy memory back

    // print answer and matrix

    return 0;
}
