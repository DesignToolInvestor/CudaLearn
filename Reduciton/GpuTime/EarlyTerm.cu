/*
  E a r l y T e r m . c u
*/

#pragma once

//#include <cuda.h>
#include "hip/hip_runtime.h"
//#include "device_launch_parameters.h"
//#include "GridHelper.cuh"

// ************************************
// This function assumes that the gird is 1D.
template<typename ElemT>
__global__ void AddReduceEarlyTerm(ElemT* partSum, ElemT* data, unsigned dataSize)
{
  // ToDo:  compare speeds with using size_t instead of unsigned
  unsigned numBlock = gridDim.x;
  unsigned threadPerBlock = blockDim.x;
  unsigned elemPerBlock = threadPerBlock * 2;

  unsigned blockNum = blockIdx.x;
  unsigned blockThread0 = blockNum * threadPerBlock;
  //unsigned blockElem0 = blockThread0 * 2;

  unsigned localThread = threadIdx.x;
  //unsigned globalThread = localThread + blockThread0;

  unsigned globalElem = 2 * blockThread0 + localThread;

  // This is the portion of the block/array that is active ... shrinks with each iteration.
  unsigned numActiveElem, numActiveThread;

  if (blockNum < numBlock - 1) {
    numActiveElem = elemPerBlock;
    numActiveThread = threadPerBlock;
  }
  else {
    numActiveElem = dataSize - 2 * blockThread0;
    numActiveThread = (numActiveElem + 1) >> 1;
  }

  // Do this thread's computation
  unsigned localCompanionElem = localThread + numActiveThread;
  if (localCompanionElem < numActiveElem) {
    unsigned otherGlobalElem = globalElem + numActiveThread;
    data[globalElem] += data[otherGlobalElem];

    __syncthreads();

    // Higher numbered threads will finish early
    unsigned count = 0;
    while (((localThread < numActiveThread) && (1 < numActiveThread)) && (count < 1)) {
      unsigned numActiveElem = numActiveThread;
      numActiveThread = (numActiveElem + 1) >> 1;

      localCompanionElem = localThread + numActiveThread;
      if (localCompanionElem < numActiveElem) {
        unsigned otherGlobalElem = globalElem + numActiveThread;
        data[globalElem] += data[otherGlobalElem];
      }
      __syncthreads();
    }
  }

  // copy partSum back
  if (localThread == 0)
    partSum[blockNum] = data[globalElem];
}

// Do instantiations
template __global__ void AddReduceEarlyTerm(float* partSum, float* data, unsigned dataSize);