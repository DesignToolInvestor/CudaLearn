#include "hip/hip_runtime.h"
﻿/*
  K e r n e l . c u
*/

// Modren C++
#include <cstdlib>
#include <iostream>

// Cuda
#include "hip/hip_runtime.h"
#include ""

using namespace std;

// ****************************************************************************
template<typename ElemT>
__global__ void KernalA(ElemT *data, unsigned size)
{
  unsigned thread = threadIdx.x;
  unsigned elem = (blockIdx.x * blockDim.x) + thread;

  data[elem] = elem;
}

// ************************************
template<typename ElemT>
__global__ void KernalB(ElemT* data, unsigned size)
{
  unsigned thread = threadIdx.x;
  unsigned elem = (blockIdx.x * blockDim.x) + thread;

  if ((elem % 2) == 0)
    data[elem] = elem;
  else
    data[elem] = -elem;
}

// ****************************************************************************
void CheckErr(hipError_t status, const char* message)
{
  if (status != hipSuccess) {
    cout << message;
    abort();
  }
}

// ************************************
// Note:  Typedef doesn't work, but using (which defines an alias) will work.
template<typename ElemT>
using ArrayInitKernT = void (*)(ElemT* data, unsigned dataElems);

template<typename ElemT, typename ArrayInitKernT<ElemT> Kern>
void InvokeKern(ElemT* data, unsigned dataElems, unsigned threadPerBlock)
{
  ElemT* data_d = NULL;

  // Choose which GPU to run on, change this on a multi-GPU system.
  CheckErr(hipSetDevice(0), "No cuda devices.");

  // Compute the grid size
  hipDeviceProp_t devProp;
  CheckErr(hipGetDeviceProperties(&devProp, 0), "Can't get device properties");

  // Allocate GPU buffers for data and partSum  
  const size_t dataBytes = dataElems * sizeof(ElemT);
  CheckErr(hipMalloc((void**)&data_d, dataBytes), "Data allocation failed");

  // Copy input vectors from host memory to GPU buffers.
  CheckErr(hipMemcpy(data_d, data, dataBytes, hipMemcpyHostToDevice), "Copying data failed");

  // Launch kernel
  unsigned numThread = dataElems;
  unsigned numBlock = (unsigned)((numThread + (threadPerBlock - 1)) / threadPerBlock);

  Kern <<<numBlock, threadPerBlock>>>(data_d, dataElems);

  // Check for any errors launching the kernel
  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    abort();
  }

  // Wait for the kernel to finish
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d\n", cudaStatus);
    abort();
  }

  // Copy data back
  CheckErr(
    hipMemcpy(data, data_d, dataBytes, hipMemcpyDeviceToHost),
    "Copy of PartSum failed");

  // Clean up
  if (data_d != NULL)
    hipFree(data_d);
}

// ****************************************************************************
int main()
{
  typedef int ElemT;

  const unsigned dataSize = 4096;
  const unsigned threadPerBlock = 256;

  ElemT data[dataSize];

  // Initialize data Method A
  // This is not sensible kernel, but it demonstrates the use of templates.
  InvokeKern<ElemT, KernalA<ElemT>>(data, dataSize, threadPerBlock);

  // Check the result
  for (unsigned i{ 0 }; i < dataSize; i++)
    if (data[i] != i)
      abort();

  cout << "Kernal A passed test.\n";

  // Initialize data Method B
  InvokeKern<ElemT, KernalB<ElemT>>(data, dataSize, threadPerBlock);

  // Check the result
  for (unsigned i{ 0 }; i < dataSize; i++)
    if ((((i % 2) == 0) && (data[i] != i)) || (((i % 2) == 1) && (data[i] != -i)))
      abort();

  cout << "Kernal B passed test.\n";

  // hipDeviceReset must be called before exiting in order for profiling and
  // tracing tools such as Nsight and Visual Profiler to show complete traces.
  CheckErr(hipDeviceReset(), "hipDeviceReset failed!");

  return 0;
}