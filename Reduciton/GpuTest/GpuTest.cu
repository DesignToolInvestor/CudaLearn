﻿/*
  G p u T e s t . c u
*/


#include <iostream>

#include <stdio.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "../Library/ReduceAdd.h"
#include "GridHelper.cuh"
#include "EarlyTerm.cuh"

using namespace std;

// ****************************************************************************
template<typename ElemT>
hipError_t ReduceAddGpu(
  ElemT& result, const ElemT* data, size_t numElem, unsigned threadPerBlock)
{
  ElemT* data_d = NULL;
  ElemT* partSum_d = NULL;
  hipError_t cudaStatus = hipSuccess;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
    goto Error;
  }

  // Compute the grid size
  hipDeviceProp_t devProp;
  cudaStatus = hipGetDeviceProperties(&devProp, 0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "Didn't get device properties!\n");
    goto Error;
  }

  unsigned numThread = (numElem - 1) / 2 + 1;
  dim3 grid = GridSizeSimple(numThread, threadPerBlock, devProp);
  unsigned numBlock = grid.x * grid.y * grid.z;

  // Allocate GPU buffers for data and partSum  
  const size_t dataBytes = numElem * sizeof(ElemT);
  cudaStatus = hipMalloc((void**) &data_d, dataBytes);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!\n");
    goto Error;
  }

  const size_t resultBytes = numBlock * sizeof(ElemT);
  cudaStatus = hipMalloc((void**)&partSum_d, resultBytes);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!\n");
    goto Error;
  }

  // Copy input vectors from host memory to GPU buffers.
  cudaStatus = hipMemcpy(data_d, data, dataBytes, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!\n");
    goto Error;
  }

  // Launch a kernel on the GPU with one thread for each element.
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  AddReduceEarlyTerm <<< grid, threadPerBlock >>> (partSum_d, data_d, numElem);

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    goto Error;
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    goto Error;
  }

  hipEventRecord(stop);
  float time = hipEventElapsedTime(&time, start,stop);
  time *= 1e3;

  cout << numElem << ", " << threadPerBlock << ", " << time << '\n';

  // Copy output vector from GPU buffer to host memory.
  ElemT* partSum = new ElemT[numBlock];
  cudaStatus = hipMemcpy(partSum, partSum_d, resultBytes, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!\n");
    goto Error;
  }
  delete[] partSum;
  
  // Print partial sum
  result = ReduceAdd(partSum, numBlock);

  // Print munged data
  ElemT mungedData[17];
  cudaStatus = hipMemcpy(mungedData, data_d, dataBytes, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!\n");
    goto Error;
  }

Error:
  if (data_d != NULL)
    hipFree(data_d);
  if (partSum_d != NULL)
    hipFree(partSum_d);

  return cudaStatus;
}

// ************************************
//int main()
//{
//  constexpr size_t minSize = 15;
//  constexpr size_t maxSize = 17;
//  constexpr unsigned threadPerBlock = 8;
//
//  for (size_t size{ minSize }; size <= maxSize; size++) {
//    int* data = new int[size];
//    for (size_t i = 0; i < size; i++)
//      data[i] = i;
//
//    int result;
//    hipError_t cudaStatus = ReduceAddGpu<int>(result, data, size, threadPerBlock);
//    delete[] data;
//
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//      fprintf(stderr, "hipDeviceReset failed!");
//      return 1;
//    }
//
//    if (result != (size - 1) * size / 2) {
//      fprintf(stderr, "Got wrong answer!");
//      return 1;
//    } else
//      fprintf(stderr, "Size = %d passed\n", size);
//  }
//
//  return 0;
//}

// Actually create something to like to
template hipError_t ReduceAddGpu<int>(
  int& result, const int* data, size_t numElem, unsigned threadPerBlock);