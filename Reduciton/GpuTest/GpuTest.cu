#include "hip/hip_runtime.h"
﻿/*
  G p u T e s t . c u
*/


#include <iostream>

#include <stdio.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "../Library/ReduceAdd.h"
#include "GridHelper.cuh"
#include "EarlyTerm.cuh"

using namespace std;

// ****************************************************************************
void Ok(hipError_t status, char* message)
{
  if (status != hipSuccess) {
    printf(message);
    abort();
  }
}

// ****************************************************************************
template<typename ElemT>
  __global__ void WarmingUp(ElemT* partSum, ElemT* data, unsigned dataSize)
{
  unsigned numBlock = blockDim.x;
  int tid = blockIdx.x * numBlock + threadIdx.x;

  partSum[tid % numBlock] = tid;
}

// ****************************************************************************
template<typename ElemT>
  void ReduceAddGpu(
    ElemT& result, const ElemT* data, size_t numElem, unsigned threadPerBlock)
{
  ElemT* data_d = NULL;
  ElemT* partSum_d = NULL;

  // Choose which GPU to run on, change this on a multi-GPU system.
  Ok(hipSetDevice(0), "No cuda devices.");

  // Compute the grid size
  hipDeviceProp_t devProp;
  Ok(hipGetDeviceProperties(&devProp, 0), "Can't get device properties");

  unsigned numThread = (numElem - 1) / 2 + 1;
  unsigned numBlock = (numThread - 1) / threadPerBlock + 1;

  // Allocate GPU buffers for data and partSum  
  const size_t dataBytes = numElem * sizeof(ElemT);
  Ok(hipMalloc((void**)&data_d, dataBytes), "Data allocation failed");

  const size_t resultBytes = numBlock * sizeof(ElemT);
  Ok(hipMalloc((void**)&partSum_d, resultBytes), "PartSum allocaiton failed");

  // Copy input vectors from host memory to GPU buffers.
  Ok(hipMemcpy(data_d, data, dataBytes, hipMemcpyHostToDevice), "Copying data failed");

  // Create timmers
  hipEvent_t preWarm, middle, postReduce;
  Ok(hipEventCreate(&preWarm), "Creation of PreWarm event failed");
  Ok(hipEventCreate(&middle), "Creation of Midle event failed");
  Ok(hipEventCreate(&postReduce), "Creation of PostReduce event failed");

  // **********************************
  // Do warmup
  Ok(hipEventRecord(preWarm), "Recording PreWarm event failed");
  WarmingUp <<< numBlock, threadPerBlock >>> (partSum_d, data_d, numElem);

  // Check for any errors launching the kernel
  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    abort();
  }

  // waits for the kernel to finish
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d\n", cudaStatus);
    abort();
  }

  // compute elapsed time
  Ok(hipEventRecord(middle), "Recording middle event failed");

  float warmTime;
  Ok(hipEventElapsedTime(&warmTime, preWarm, middle), "Warmup time failed.");
  warmTime *= 1e-3;

  // **********************************
  // Do Add Reduce
  AddReduceEarlyTerm << < numBlock, threadPerBlock >> > (partSum_d, data_d, numElem);

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    abort();
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d\n", cudaStatus);
    abort();
  }

  // Deal with time
  Ok(hipEventRecord(postReduce), "Recording Stop event failed");

  float reduceTime;
  Ok(hipEventElapsedTime(&reduceTime, middle, postReduce),"reduce time feaild");
  reduceTime *= 1e-3;

  cout << numElem << ", " << threadPerBlock << ", " << warmTime << ", " << reduceTime << '\n';

  // Copy output vector from GPU buffer to host memory.
  ElemT* partSum = new ElemT[numBlock];
  Ok(
    hipMemcpy(partSum, partSum_d, resultBytes, hipMemcpyDeviceToHost), 
    "Copy of PartSum failed");

  result = ReduceAdd(partSum, numBlock);
  delete[] partSum;

  // Clean up
  if (data_d != NULL)
    hipFree(data_d);
  if (partSum_d != NULL)
    hipFree(partSum_d);
}

// ************************************
// Actually create something to like to
template void ReduceAddGpu<int>(
  int& result, const int* data, size_t numElem, unsigned threadPerBlock);