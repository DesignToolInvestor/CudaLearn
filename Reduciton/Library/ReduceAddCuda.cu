#include "hip/hip_runtime.h"
/*
  R e d u c e A d d C u d a . c u
*/

#include <iostream>

#include <stdio.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "../Library/EarlyTerm.cuh"
//#include "../Library/ReduceAdd.h"
#include "../Library/TimerWrap.cuh"

using namespace std;

// ****************************************************************************
// ToDo:  This is a very kludgey work around.  Temples are not working, only in this case.
float ReduceAdd(const float* data, size_t numElem)
{
  float result = 0;
  for (unsigned i = 0; i < numElem; i++)
    result += data[i];

  return result;
}

int ReduceAdd(const int* data, size_t numElem)
{
  int result = 0;
  for (unsigned i = 0; i < numElem; i++)
    result += data[i];

  return result;
}

// ****************************************************************************
template<typename ElemT>
__global__ void WarmUp_d(ElemT* partSum, ElemT* inArray, unsigned dataSize)
{
  unsigned numBlock = blockDim.x;
  int tid = (blockIdx.x * numBlock) + threadIdx.x;

  if ((tid % 2) == 0)
    inArray[tid] = tid;
  else
    inArray[tid] = tid;

  if (tid < numBlock)
    partSum[tid] = tid;
}

// ****************************************************************************
void CheckErr(hipError_t status, const char* message)
{
  if (status != hipSuccess) {
    printf(message);
    abort();
  }
}

// ************************************
template<typename ElemT>
void WarmUp(
  ElemT* outArray_d, ElemT* inArray_d, size_t numElems,
  unsigned numBlock, unsigned threadPerBlock)
{
  // launch kernel
  WarmUp_d<ElemT> << < numBlock, threadPerBlock >> > (outArray_d, inArray_d, numElems);

  // Check for any errors launching the kernel
  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    abort();
  }

  // Wait for the kernel to finish
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d\n", cudaStatus);
    abort();
  }
}

// ************************************
template<typename ElemT>
void AddReduceEarlyTerm(
  ElemT* outArray_d, ElemT* inArray_d, size_t numElems,
  unsigned numBlock, unsigned threadPerBlock)
{
  // Launch device code
  AddReduceEarlyTerm<ElemT> << < numBlock, threadPerBlock >> > (outArray_d, inArray_d, numElems);

  // Check for any errors launching the kernel
  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    abort();
  }

  // Wait for the kernel to finish
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d\n", cudaStatus);
    abort();
  }
}

// ****************************************************************************
template<typename ElemT>
void ReduceAddCuda(
  ElemT& result, const ElemT* inArray, size_t origNumElem, unsigned threadPerBlock)
{
  ElemT* inArray_d = NULL;
  ElemT* outArray_d = NULL;

  // Choose which GPU to run on, change this on a multi-GPU system.
  CheckErr(hipSetDevice(0), "No cuda devices.");

  // Check if the kernel should be called at all.
  if (origNumElem < 2 * threadPerBlock)
    result = ReduceAdd(inArray, origNumElem);
  else {
    // Start the clock
    TickCountT startTicks = ReadTicks_d();

    // Allocate GPU buffers for inArray
    const size_t dataBytes = origNumElem * sizeof(ElemT);
    CheckErr(hipMalloc((void**)&inArray_d, dataBytes), "Data allocation failed");

    // Copy inArray from host memory to GPU.
    CheckErr(hipMemcpy(inArray_d, inArray, dataBytes, hipMemcpyHostToDevice), "Copying inArray failed");

    // Recurse until the result is small enough to do in the CPU
    size_t numElems = origNumElem;
    unsigned numBlock;
    size_t outBytes;

    TickCountT stopTicks[10] = { 0 };
    unsigned level = 0;

    while (2 * threadPerBlock <= numElems) {
      // Compute launch parameters
      unsigned numThread = (unsigned)((numElems + 1) >> 1);
      numBlock = (unsigned)((numThread + threadPerBlock - 1) / threadPerBlock);

      // Allocate the output array
      outBytes = numBlock * sizeof(ElemT);
      CheckErr(hipMalloc((void**)&outArray_d, outBytes), "Result allocation failed");

      // Launch the kernel and wait for synchronization
      AddReduceEarlyTerm(outArray_d, inArray_d, numElems, numBlock, threadPerBlock);
      stopTicks[level++] = ReadTicks_d();

      // Do double-buffering thing
      CheckErr(hipFree(inArray_d), "CudaFree failed.");
      inArray_d = outArray_d;
      numElems = numBlock;
    }

    // Copy output vector from GPU buffer to host memory.
    ElemT* outArray = new ElemT[numBlock];
    CheckErr(
      hipMemcpy(outArray, outArray_d, outBytes, hipMemcpyDeviceToHost),
      "Copy of OutArray failed");
    CheckErr(hipFree(inArray_d), "CudaFree failed.");

    // Add using CPU
    result = ReduceAdd(outArray, numBlock);
    delete[] outArray;

    // Stop the clock
    printf("%lld, %d, ", origNumElem, threadPerBlock);
    for (unsigned i{ 0 }; i < level; i++)
      printf(", % f", TicksToSecs_d(stopTicks[i] - startTicks));
    printf("\n");
  }
}
