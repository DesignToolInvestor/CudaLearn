/*
  D e v I n f o . c u
*/

#pragma once

#include <cstdio>
#include "hip/hip_runtime.h"
#include "devInfo.cuh"

// ****************************************************************************
// ToDo:  Move to a separate file at some point

void DevInfo::CheckOk(const hipError_t status)
{
  // ToDo:  Want to halt the debugger, but need to decide on an exception pattern
  if (status != hipSuccess)
    abort();
}

// ************************************************************
DevInfo::DevInfo() {
  constexpr unsigned devNum = 0;

  // Initialize
  CheckOk(hipSetDevice(devNum));
  
  // Get number of GPUs
  int temp;
  CheckOk(hipGetDeviceCount(&temp));
  if (temp < 0)
    abort();
  numDev = (unsigned)temp;

  // Get device properties
  hipDeviceProp_t devProp;
  CheckOk(hipGetDeviceProperties(&devProp, 0));

  // Get stuff in devProp
  numSm = devProp.multiProcessorCount;
  compClassMajor = devProp.major;
  compClassMinor = devProp.minor;

  maxThreadPerBlock = devProp.maxThreadsPerBlock;
  maxThreadPerSm = devProp.maxThreadsPerMultiProcessor;
  maxBlockPerSm = devProp.maxBlocksPerMultiProcessor;

  // F32 cores per SM
  typedef struct {
    int major, minor, f32Cores;
  } CoreInfoT;

  constexpr unsigned numVer = 17;
  CoreInfoT smInfo[numVer] = {
    {3,0, 192}, {3,2, 192}, {3,5, 192}, {3,7, 192},
    {5,0, 128}, {5,2, 128}, {5,3, 128},
    {6,0,  64}, {6,1, 128}, {6,2, 128},
    {7,0,  64}, {7,2,  64}, {7,5,  64},
    {8,0,  64}, {8,6, 128}, {8,9, 128},
    {9,0, 128} };

  int i = 0;
  while ((i < numVer) && ((smInfo[i].major != compClassMajor) || (smInfo[i].minor != compClassMinor)))
    i++;

  if (i == numVer)
    abort();

  numF32CorePerSm = smInfo[i].f32Cores;
}

// ************************************************************
// Accessor functions
unsigned DevInfo::NumDev() const
{
  return numDev;
}

unsigned DevInfo::NumSm() const
{
  return numSm;
}

unsigned DevInfo::CompClassMajor() const
{
  return compClassMajor;
}

unsigned DevInfo::CompClassMinor() const
{
  return compClassMinor;
}

unsigned DevInfo::NumF32CorePerSm() const
{
  return numF32CorePerSm;
};

unsigned DevInfo::MaxThreadPerBlock() const
{
  return maxThreadPerBlock;
}

unsigned DevInfo::MaxThreadPerSm() const
{
  return maxThreadPerSm;
}

unsigned DevInfo::MaxBlockPerSm() const
{
  return maxBlockPerSm;
}
