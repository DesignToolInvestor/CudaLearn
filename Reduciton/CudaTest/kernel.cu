﻿
#include "hip/hip_runtime.h"
//#include "device_launch_parameters.h"

#include <stdio.h>


int main()
{
  hipError_t cudaStatus;
  constexpr unsigned devNum = 0;

  cudaStatus = hipSetDevice(devNum);
  if (cudaStatus != hipSuccess)
    fprintf(stderr, "hipSetDevice failed!");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, devNum);

  // cudaDeviceReset must be called before exiting in order for profiling and tracing tools (such 
  // as Nsight and Visual Profiler) to show complete traces.
  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;
  }

  return 0;
}
