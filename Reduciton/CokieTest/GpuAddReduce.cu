#include "hip/hip_runtime.h"
#include "GpuAddReduce.h"
#include "UtilMiscCokie.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stddef.h>

#include <format>
#include <iostream>
#include <random>


__global__ void AddReduceKernel(float* g_idata, float* g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;

    //covnert global data pointer to the local pointer
    // of this block
    float* block = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (tid >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (((tid % (2 * stride)) == 0) && ((tid + stride) < blockDim.x)) {
            block[tid] += block[tid + stride];
        }

        //debug
        /*if (tid == 0) {
            printf("stride: %d , blockSize: %d\n", stride, blockDim.x);
        }*/

        // synchronize within block
        __syncthreads();
    }

    //write result for this block to global mem
    if (tid == 0) {
        g_odata[blockIdx.x] = block[0];

        //debug
        /*printf("thread %d result: %f \n", tid, *block);*/
    }
}

template<typename ElemT>
hipError_t ReduceAddGpu(const ElemT* data, int dataSize, ElemT& result)
{
    ElemT* data_d = NULL;
    ElemT* partSum_d = NULL;
    hipError_t cudaStatus = hipSuccess;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        goto Error;
    }

    // Compute gird parameters
    const unsigned elemPerBlock = 1536/2;
    const unsigned numBlock = ((dataSize - 1) / elemPerBlock) + 1;
    const unsigned threadPerBlock = elemPerBlock;

    // Allocate GPU buffers for data and partSum  
    const size_t dataBytes = dataSize * sizeof(ElemT);
    cudaStatus = hipMalloc((void**)&data_d, dataBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }

    const size_t resultBytes = numBlock * sizeof(ElemT);
    cudaStatus = hipMalloc((void**)&partSum_d, resultBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(data_d, data, dataBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

    // timing code
    TickCountT start_ticks = ReadTicks();

    // Launch a kernel on the GPU with one thread for each element.
    AddReduceKernel << < numBlock, threadPerBlock >> > (data_d, partSum_d, dataSize);

    // timing code
    hipDeviceSynchronize();
    TickCountT end_ticks = ReadTicks();
    float time_elapsed = TicksToSecs(end_ticks - start_ticks);
    printf("%f %d\n", time_elapsed, dataSize);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    ElemT* partSum = new ElemT[numBlock];
    cudaStatus = hipMemcpy(partSum, partSum_d, resultBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

    // Print partial sum
    result = ReduceAdd(partSum, numBlock);

    // Print munged data
    ElemT* mungedData = new float[dataSize];
    cudaStatus = hipMemcpy(mungedData, data_d, dataBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Error;
    }

    // debug
    /*std::cout << "\n";
    for (int i = 0; i < dataSize; i++) {
        std::cout << mungedData[i] << ", ";
    }
    std::cout << "\n";*/

Error:
    if (data_d != NULL)
        hipFree(data_d);
    if (partSum_d != NULL)
        hipFree(partSum_d);

    return cudaStatus;
}

template hipError_t ReduceAddGpu<float>(const float* data, int dataSize, float& result);