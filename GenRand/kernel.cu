﻿/*
  G e n R a n d . c u
*/

#include "hip/hip_runtime.h"


//#include <stdio.h>

// C++ files
#include <iostream>

using namespace std;

// ************************************
// This is the turbo pascal random number generator.  It is implicitly mod 2^32.
inline __device__ unsigned NextSeedDev(unsigned seed)
{
  const unsigned A = 134'775'813;
  const unsigned B = 1;

  return (A * seed + B);
}

inline unsigned NextSeed(unsigned seed)
{
  const unsigned A = 134'775'813;
  const unsigned B = 1;

  return (A * seed + B);
}

// ************************************
__global__ void RandGenKern(unsigned* results, unsigned* threadSeed, int randPerThread)
{
  int thread = threadIdx.x;

  unsigned rowStart = randPerThread * thread;
  unsigned seed = threadSeed[thread];

  for (int i = 0; i < randPerThread; i++)
    seed = results[rowStart + i] = NextSeedDev(seed);
}

// ************************************
hipError_t RandGenLaunch(
  unsigned* result, const unsigned masterSeed, const int numThread, const int randPerThread)
{
  const unsigned numResult = numThread * randPerThread;

  // Need to be declared here because of the goto's
  unsigned* threadSeed = new unsigned[numThread];
  unsigned seed;

  // Verify that we can run on the first CPU 
  // ToDo:  Fix so that it does something sensible on a multi-GPU system.
  hipError_t cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // Allocate space on the GPU
  unsigned* resultDev;
  cudaStatus = hipMalloc((void**)&resultDev, numResult * sizeof(unsigned));
  
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  unsigned* threadSeedDev;
  cudaStatus = hipMalloc((void**)&threadSeedDev, numThread * sizeof(unsigned));
  
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  // Generate thread seeds ... On the CPU, because it's probably not faster on the GPU.
  seed = threadSeed[0] = masterSeed;
  for (int i{ 1 }; i < numThread; i++)
    seed = threadSeed[i] = NextSeed(seed);

  // Copy thread seed to device
  cudaStatus = hipMemcpy(
    threadSeedDev, threadSeed, numThread * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  // Launch a kernel on the GPU with one thread for each element.
  RandGenKern <<<1, numThread>>> (resultDev, threadSeedDev, randPerThread);

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    goto Error;
  }

  // cudaDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    goto Error;
  }

  // Copy output vector from GPU buffer to host memory.
  cudaStatus = hipMemcpy(result, resultDev, numResult * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

Error:
  if (resultDev)
    hipFree(resultDev);
  if (threadSeedDev)
    hipFree(threadSeedDev);

  return cudaStatus;
}

// ************************************
int main()
{
  // core parameters
  const int numThread = 100;
  const int randPerThread = 100;
  const unsigned masterSeed = 4;

  // Derived constants
  const int numResult = numThread * randPerThread;
  unsigned result[numResult] = { 0 };

  // call kernel
  hipError_t status = RandGenLaunch(result, masterSeed, numThread, randPerThread);
  if (status == hipSuccess)
    cout << "No error\n";
  else
    cout << "Error code = " << status << "\n";

  return 0;
}